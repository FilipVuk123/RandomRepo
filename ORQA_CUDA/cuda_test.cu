#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

extern "C" {
	#include "orqa_clock.h"
}

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1000000;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Run kernel on 1M elements on the GPU
	orqa_clock_t clock = orqa_time_now();
	add<<<1, 256>>>(N, x, y);
	printf("Time passed: %f\n",orqa_get_time_diff_msec(clock, orqa_time_now()));
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	for (int i = 0; i < 10; ++i)
		printf("vector[%d] = %f\n", i, y[i]);

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}