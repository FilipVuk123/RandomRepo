#include <iostream>
#include <time.h>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;
__host__ __device__ unsigned char bgr2y(int R, int G, int B){
  int Y = ((66 * R + 129 * G + 25 * B + 128) >> 8) + 16;
  return (unsigned char)((Y<0)? 0 : ((Y > 255) ? 255 : Y));}
__host__ __device__ int bgr2u(int R, int G, int B){
  int U = ((-38 * R - 74 * G + 112 * B + 128) >> 8) + 128;
  return (unsigned char)((U<0)? 0 : ((U > 255) ? 255 : U));}
__host__ __device__ int bgr2v(int R, int G, int B){
  int V = ((112 * R - 94 * G - 18 * B + 128) >> 8) + 128;
  return (unsigned char)((V<0)? 0 : ((V > 255) ? 255 : V));}

void bgr_to_yuv420sp(unsigned  char* yuv420p, unsigned char* bgr, int width, int height)
{
    if (yuv420p == NULL || bgr== NULL)
        return;
    int frameSize = width*height;

    int yIndex = 0;
    int uIndex = frameSize;

    int R, G, B;
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            B = bgr[(i * width + j) * 3 + 0];
            G = bgr[(i * width + j) * 3 + 1];
            R = bgr[(i * width + j) * 3 + 2];

            //BGR to YUV
            yuv420p[yIndex++] = bgr2y(R,G,B);
            if (i % 2 == 0 && j % 2 == 0)
            {
                yuv420p[uIndex] = bgr2u(R,G,B);
                yuv420p[uIndex+1] = bgr2v(R,G,B);
                uIndex+=2;
            }
        }
    }
}
void bgr_to_yuv420p(unsigned  char* yuv420p, unsigned char* bgr, int width, int height)
{
    if (yuv420p == NULL || bgr== NULL)
        return;
    int frameSize = width*height;
    int chromaSize = frameSize / 4;

    int yIndex = 0;
    int uIndex = frameSize;
    int vIndex = frameSize + chromaSize;

    int R, G, B, Y, U, V;
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            B = bgr[(i * width + j) * 3 + 0];
            G = bgr[(i * width + j) * 3 + 1];
            R = bgr[(i * width + j) * 3 + 2];

            //BGR to YUV
            Y = ((66 * R + 129 * G + 25 * B + 128) >> 8) + 16;
            U = ((-38 * R - 74 * G + 112 * B + 128) >> 8) + 128;
            V = ((112 * R - 94 * G - 18 * B + 128) >> 8) + 128;

            yuv420p[yIndex++] = (unsigned char)((Y < 0) ? 0 : ((Y > 255) ? 255 : Y));
            if (i % 2 == 0 && j % 2 == 0)
            {
                yuv420p[uIndex++] = (unsigned char)((U < 0) ? 0 : ((U > 255) ? 255 : U));
                yuv420p[vIndex++] = (unsigned char)((V < 0) ? 0 : ((V > 255) ? 255 : V));
            }
        }
    }
}
//kernel function to convert bgr to yuv420sp
__global__ void bgr2yuv420sp(uchar3 *  d_in, unsigned char * d_out,
                               uint imgheight, uint imgwidth)
{

    int col_num = blockIdx.x*blockDim.x+threadIdx.x;
    int row_num = blockIdx.y*blockDim.y+threadIdx.y;

    if ((row_num < imgheight) && (col_num < imgwidth))
    {
//        uint32_t a = *((uint32_t *)&dinput[global_offset*3]);
        int global_offset = row_num*imgwidth+col_num;

        int r,g,b;
        r = int(d_in[global_offset].z);
        g = int (d_in[global_offset].y);
        b = int (d_in[global_offset].x);


        d_out[row_num * imgwidth + col_num] = bgr2y(r,g,b);
        if(((threadIdx.x & 1) == 0)  && ((threadIdx.y & 1) == 0)){
            int uv_offset = imgwidth*imgheight+((row_num>>1)*imgwidth)+col_num;
            d_out[uv_offset] = bgr2u(r,g,b);
            d_out[uv_offset+1] = bgr2v(r,g,b);

        }

    }
}
//kernel function to convert bgr to yuv420p
__global__ void bgr2yuv420p(uchar3 *  d_in, unsigned char * d_out,
                               uint imgheight, uint imgwidth)
{

    int col_num = blockIdx.x*blockDim.x+threadIdx.x;
    int row_num = blockIdx.y*blockDim.y+threadIdx.y;

    if ((row_num < imgheight) && (col_num < imgwidth))
    {
//        uint32_t a = *((uint32_t *)&dinput[global_offset*3]);
        int global_offset = row_num*imgwidth+col_num;

        int r,g,b;
        r = int(d_in[global_offset].z);
        g = int (d_in[global_offset].y);
        b = int (d_in[global_offset].x);


        d_out[row_num * imgwidth + col_num] = bgr2y(r,g,b);
        if(((threadIdx.x & 1) == 0)  && ((threadIdx.y & 1) == 0)){
            int u_offset = imgwidth*imgheight+((row_num>>1)*(imgwidth>>1))+(col_num>>1);
            d_out[u_offset] = bgr2u(r,g,b);
            int v_offset = u_offset+((imgheight>>1)*(imgwidth>>1));
            d_out[v_offset] = bgr2v(r,g,b);

        }
    }
}


int main(void)
{

    const uint imgheight = 1000;
    const uint imgwidth = 1500;

    //input and output
    uchar3 *d_in;
    unsigned char *d_out;
    uchar3 *idata = new uchar3[imgheight*imgwidth];
    unsigned char *odata = new unsigned char[imgheight*imgwidth*3/2];
    unsigned char *cdata = new unsigned char[imgheight*imgwidth*3/2];
    uchar3 pix;
    for (int i = 0; i < imgheight*imgwidth; i++){
      pix.x = (rand()%30)+40;
      pix.y = (rand()%30)+40;
      pix.z = (rand()%30)+40;
      idata[i] = pix;}
    for (int i = 0; i < imgheight*imgwidth; i++) idata[i] = pix;
    bgr_to_yuv420p(cdata, (unsigned char*) idata, imgwidth, imgheight);
    // malloc memo in gpu
    hipMalloc((void**)&d_in, imgheight*imgwidth*sizeof(uchar3));
    hipMalloc((void**)&d_out, imgheight*imgwidth*sizeof(unsigned char) * 3 / 2);

    //copy image from cpu to gpu
    hipMemcpy(d_in, idata, imgheight*imgwidth*sizeof(uchar3), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    //run kernel function
    bgr2yuv420p<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, imgheight, imgwidth);

    hipDeviceSynchronize();

    //copy yuv420p from gpu to cpu
    hipMemcpy(odata, d_out, imgheight*imgwidth*sizeof(unsigned char) * 3 / 2, hipMemcpyDeviceToHost);
    for (int i = 0; i < (imgwidth*imgheight*3/2); i++) if (odata[i] != cdata[i]) {std::cout << "mismatch at: " << i << " was: " << (int)odata[i] << " should be: " << (int)cdata[i] << std::endl; return 0;}
    hipFree(d_in);
    hipFree(d_out);


    return 0;

}